#include "hip/hip_runtime.h"
#include "Vec.h"
#include "Ray.h"
#include "Sphere.h"
#include "Traced.h"
#include "Scene.h"
#include <iostream>
#include <cmath>
#include <chrono>
#include <vector>
#include <algorithm>
#include <iterator>

#define WIDTH  800
#define HEIGHT 800

//Check intersection of ray and sphere, solve for t
__device__
float intersect_sphere(const Sphere & s, const Ray & r) {
    Vec Ac = r.A() - s.c();
    float a = dot(r.B(), r.B());
    float b = 2 * dot(r.B(), Ac);
    float c = dot(Ac, Ac) - s.r() * s.r();
    float disc = b * b - 4 * a * c;
    if(disc > 0) {
        float dist_sqrt = sqrt(disc);
        float q = b < 0 ? (-b - dist_sqrt) / 2.0 : (-b + dist_sqrt) / 2.0;
        float t0 = q / a;
        float t1 = c / q;
        float temp = t0;
        t0 = std::min(t0, t1);
        t1 = std::max(temp, t1);
        if(t1 >= 0) return t0 < 0 ? t1 : t0;
    }
    return INFINITY;
}

__device__
Traced trace_ray(Ray & r, Sphere * scene, Vec LIGHT) {
    float t = INFINITY;
    float t_object;
    int object_i = 0;
    
    for(int i = 0; i < 4; ++i) {
        t_object = intersect_sphere(scene[i], r);
        if(t_object < t) {
            t = t_object;
            object_i = i;
        }
    }
    if(t == INFINITY) {
        return Traced();
    }    
    Sphere object = scene[object_i];
    Vec M = r.P(t);
    Vec N = norm(M - object.c());
    Vec toL = norm(LIGHT - M);
    Vec toO = norm(r.A() - M);
    float l[3];
    int j = 0;
    for(int i = 0; i < 4; ++i) {
        if(i != object_i) {
            l[j] = intersect_sphere(object, Ray(M + 0.0001 * N, toL));
            ++j;
        }
    }    
    if(sizeof(l)/sizeof(*l) != 0 && *std::min_element(std::begin(l), std::end(l)) < INFINITY) {
        return Traced();
    }
    Vec col = Vec(0.05,0.05,0.05);
    // Shading osv.
    col += 1 * std::max(dot(N, toL), 0.0f) * object.col();
    col += pow((1 * std::max(dot(N, norm(toL + toO)), 0.0f) * Vec(1,1,1)), 50);
    return Traced(object, M, N, col);
}

__host__
float clip(float f) {
    if(f < 0.0) {
        return 0.0;
    }    
    if(f > 1.0) {
        return 1.0*255.999;
    }
    return f*255.999;
}

__global__
void run(Vec * res, Sphere * scene, Vec LIGHT) {
    Vec O = Vec(0,0,2); // Camera position
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if(i >= WIDTH || j >= HEIGHT) return;
    int index = j*WIDTH + i;
    Vec col; 
    float I = -1.0 + (2.0*i/(WIDTH-1.0));
    float J = -1.0 + (2.0*j/(HEIGHT-1.0));
    Vec D = norm(Vec(I,J,0) - O);
    Vec rayO = O;
    Vec rayD = D;
    int depth = 0;
    float ref = 1;
    while(depth < 5) {
        Ray OD = Ray(rayO, rayD);
        Traced traced = trace_ray(OD, scene, LIGHT);
        if(traced.m_col_ray.x() == -1 && traced.m_col_ray.y() == -1 && traced.m_col_ray.z() == -1) {
            break; 
        }
        Vec M = traced.m_M;
        Vec N = traced.m_N; 
        Vec col_ray = traced.m_col_ray;
        rayO = M + 0.0001 * N;
        rayD = norm(rayD - 2 * dot(rayD, N) * N);
        col += ref * col_ray;
        ref *= traced.m_sphere.ref();
        ++depth;
    }
    res[index] = col;
}

int main() {
    // Setup
    Sphere * scene = makeScene(0);
    Vec * res;
    Vec LIGHT = Vec(-5,-5,10);
    int blocks_x = 8;
    int blocks_y = 8;
    int N = HEIGHT * WIDTH;
    hipMallocManaged(&res, N*sizeof(Vec));
    // Denna är lite spännande
    dim3 blocks(WIDTH/blocks_x+1, HEIGHT/blocks_y+1);
    dim3 threads(blocks_x, blocks_y);
    

    auto start = std::chrono::system_clock::now();
    
    run<<<blocks,threads>>>(res, scene, LIGHT);
    hipDeviceSynchronize();

    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cerr << "GPU time: " << elapsed.count() << " seconds" << std::endl;
    

    // Pipe to file
    
    std::cout << "P3\n" << WIDTH << ' ' << HEIGHT << "\n255\n";
    for(int i = 0; i < WIDTH; ++i) {
        for(int j = HEIGHT - 1; j >= 0; --j) {
            int index = j*WIDTH + i;
            std::cout << clip(res[index].x()) << ' ' << clip(res[index].y()) << ' ' << clip(res[index].z()) << '\n';
        }
    }
    
    // Cleanup
    hipFree(scene);
    hipFree(res);

    return 0;
}
